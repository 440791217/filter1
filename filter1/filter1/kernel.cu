#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <test.h>
#include <filter_sys.h>
#include <Reader.h>
#include <debuglog.h>
#include <Logger.h>
#include <GpuFilter.h>
#include <Detector.h>
using namespace std;

__global__ void test(FilterSys filterSys);
bool debug = true;
bool print2local = false;

Logger logger;
int main()
{
	char tail_mode = GPU_FILTER_TMODE_16_16, window_mode = GPU_FILTER_WMODE_5_5;
	char inject_mode = GPU_FILTER_INJECT_KN, inject_mode_extension = GPU_FILTER_INJECT_EX_K1;
	char rundancy = 2, inejct_channels = 1,filter_mode=GPU_FILTER_MODE_GAUSS;
	char pic_mode = READER_MODE_TEST;
	int pic_test_image = READER_TEST_IMAGE_256_256;
	INT16 *dst_arr[10];
	INT16 **dst_arr2d[10];
	INT16 *check_arr;
	INT32 frame_datasize;
	struct_gpu_filter_info gpu_filter_info_arr[10];
	struct_detect_info detect_info;

	if (window_mode == GPU_FILTER_WMODE_3_3) {

	}
	else if (window_mode == GPU_FILTER_WMODE_5_5) {

	}
	else if (window_mode == GPU_FILTER_WMODE_7_7) {

	}
	else if (window_mode == GPU_FILTER_WMODE_ACROSS) {

	}
	else {
		Msg_Info("invalid window mode!\r\n");
		return COMMON_RC_ERR;
	}

	if (tail_mode == GPU_FILTER_TMODE_8_8) {

	}
	else if (tail_mode == GPU_FILTER_TMODE_16_16) {

	}
	else if (tail_mode == GPU_FILTER_TMODE_32_32) {

	}
	else {
		Msg_Info("invalid tail mode!\r\n");
		return COMMON_RC_ERR;
	}

	if (inject_mode == GPU_FILTER_INJECT_NONE) {

	}
	else if (inject_mode == GPU_FILTER_INJECT_KN) {

	}
	else {
		Msg_Info("invalid inject mode!\r\n");
		return COMMON_RC_ERR;
	}
	if (inejct_channels >= rundancy) {
		Msg_Info("invalid inject channels!\r\n");
		return COMMON_RC_ERR;
	}
	logger.init("simulation", "log");
	Reader reader;
	//init test matrix
	struct_pic_info pic_info;
	pic_info.height = 256;
	pic_info.width = 256;
	pic_info.value_range = 256;
	if (pic_mode == READER_MODE_TEST) {
		pic_info.mode = READER_MODE_TEST;
		pic_info.test_image_size = READER_TEST_IMAGE_256_256;
		reader.read_data(NULL, &pic_info);
	}

	if (print2local) {
		logger.write("random matrix");
		logger.br();
		for (int i = 0; i < pic_info.height; i++) {
			for (int j = 0; j < pic_info.width; j++) {
				INT16 value = pic_info.data2d[i][j];
				char buffer[10];
				sprintf(buffer, "%3d", value);
				logger.write(buffer);
				logger.write(",");
			}
			logger.br();
		}
	}
	//return 0;

	frame_datasize = pic_info.width*pic_info.height;
	for (int i = 0; i < rundancy; i++) {
		GpuFilter filter;
		struct_gpu_filter_info *gpu_filter_info = &gpu_filter_info_arr[i];
		gpu_filter_info->width = pic_info.width;
		gpu_filter_info->height = pic_info.height;
		gpu_filter_info->src2d = (INT16**)malloc(sizeof(INT16*)*pic_info.height);
		gpu_filter_info->dst2d = (INT16**)malloc(sizeof(INT16*)*pic_info.height);
		for (int j = 0; j < gpu_filter_info->height; j++) {
			gpu_filter_info->src2d[j] = (INT16*)malloc(sizeof(INT16)*gpu_filter_info->width);
			gpu_filter_info->dst2d[j] = (INT16*)malloc(sizeof(INT16)*gpu_filter_info->width);
			memcpy(gpu_filter_info->src2d[j], pic_info.data2d[j], gpu_filter_info->width * sizeof(INT16));
		}
		gpu_filter_info->filter_mode = GPU_FILTER_MODE_MID;
		gpu_filter_info->value_range = pic_info.value_range;
		gpu_filter_info->filter_mode = filter_mode;
		gpu_filter_info->window_mode = window_mode;
		gpu_filter_info->tail_mode = tail_mode;
		gpu_filter_info->inject_mode_extension = inject_mode_extension;
		gpu_filter_info->rundancy = rundancy;

		if (i < inejct_channels) {
			gpu_filter_info->inject_mode = inject_mode;
		}
		else {
			gpu_filter_info->inject_mode = GPU_FILTER_INJECT_NONE;
		}
		int rc = filter.run_gpu(gpu_filter_info);
		dst_arr2d[i] = gpu_filter_info->dst2d;
		Msg_Info("filter%d rc:%d\r\n", i, rc);
	}
	return 0;


	for (int i = 0; i < rundancy; i++) {
		struct_gpu_filter_info *gpu_filter_info = &gpu_filter_info_arr[i];
		detect_info.dst[i] = gpu_filter_info->dst;
		detect_info.frame_datasize = frame_datasize;
	}


	if (print2local) {
		logger.write("filtered matrix");
		logger.br();
		for (int i = 0; i < pic_info.height; i++) {
			for (int j = 0; j < pic_info.width; j++) {
				int pos = i * pic_info.width + j;
				INT16 value = *(dst_arr[0] + pos);
				char buffer[10];
				sprintf(buffer, "%3d", value);
				logger.write(buffer);
				logger.write(",");
			}
			logger.br();
		}

	}

	Detector detector;
	detector.dmr_detect(&detect_info);
	{
		string errmode="null";
		if (inject_mode == GPU_FILTER_INJECT_KN) {
			if (inject_mode_extension == GPU_FILTER_INJECT_EX_K1) {
				errmode = "K1";
			}
			else if (inject_mode_extension == GPU_FILTER_INJECT_EX_K2) {
				errmode = "K2";
			}
			else if (inject_mode_extension == GPU_FILTER_INJECT_EX_K3) {
				errmode = "K3";
			}

		}
		string window_info = "";
		if (window_mode == GPU_FILTER_WMODE_ACROSS) {
			window_info = "across_window,item_counts:5,";
		}
		else if (window_mode == GPU_FILTER_WMODE_3_3) {
			window_info = "3_3_window,item_counts:9,";
		}
		else if (window_mode == GPU_FILTER_WMODE_5_5) {
			window_info = "5_5_window,item_counts:25,";
		}
		logger.write("filter info");
		logger.br();
		string msg1 = "height:"+to_string(pic_info.height)+",width:"+to_string(pic_info.width)+",errormode:"+errmode+",window_info:"+window_info;
		logger.write(msg1);
		logger.br();
		Msg_Info("%s\r\n", msg1.c_str());
		logger.write("detect arr");
		logger.br();
		string msg2 ="total pixels:"+to_string(detect_info.frame_datasize)+",positive nums:"+to_string(detect_info.num_not_equal)+",negative:"+to_string(detect_info.num_equal)+ string(",pecentage equal:") + to_string(detect_info.pecentage_equal) + ",not equal:" + to_string(detect_info.pecentage_not_equal);
		logger.write(msg2);
		logger.br();
		Msg_Info("%s\r\n", msg2.c_str());
	}

	for (int i = 0; i < rundancy; i++) {
		struct_gpu_filter_info *gpu_filter_info = &gpu_filter_info_arr[i];
		free(gpu_filter_info->src);
		free(gpu_filter_info->dst);
	}

	reader.destory(&pic_info);

	return 0;
}

__global__ void test(FilterSys filterSys) {
	printf("%d\r\n", filterSys.FRAME_WIDTH);
}

